
#include <hip/hip_runtime.h>
/*==========================================================
 * patch2hank.cu
 *
 * making block hankel matrix
 *
 * compile command
 * nvcc('patch2hank_single.cu -arch sm_35')
 *
 * This is a MEX-file for MATLAB.
 *
 *========================================================*/
/* $created: 11-Mar-2015 $ */

// #define fmin(a,b) ((a) < (b) ? (a) : (b))
// #define fmax(a,b) ((a) > (b) ? (a) : (b))
// #include "mex.h"
// #include "cuda.h"

__global__ void patch2hank_single(float* out,float* y,int sy,int sx,int sz,int firy, int firx)
{
    
    int idx = threadIdx.x+blockIdx.x*blockDim.x;
    int um=sy-firy+1;
    int un=firy;
    int ii=0,jj=0,zz=0,di=0,dj=0,ri=0,rj=0,pi=0,pj=0,k=0;
    if ( idx < (sy-firy+1)*(sx-firx+1)*firy*firx*sz )
    {
        zz=(int)(idx/((sy-firy+1)*(sx-firx+1)*firy*firx));
        k=idx%((sy-firy+1)*(sx-firx+1)*firy*firx);
        ii=k%((sy-firy+1)*(sx-firx+1));
        jj=(int)(k/((sy-firy+1)*(sx-firx+1)));        
        
        di=ii/um;
        dj=jj/un;
        
        ri=ii%um;
        rj=jj%un;
        
        pi= ri+rj;
        pj= di+dj;
        
        out[ii + jj*((sy-firy+1)*(sx-firx+1)) + zz*((sy-firy+1)*(sx-firx+1)*firy*firx)]=y[pi+pj*sy+zz*sy*sx];
    }
}