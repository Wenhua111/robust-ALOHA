
#include <hip/hip_runtime.h>
/*==========================================================
 * patch2hank.cu
 *
 * making block hankel matrix
 *
 * compile command
 * nvcc('hank2patch_single.cu -arch sm_35')
 *
 * This is a MEX-file for MATLAB.
 *
 *========================================================*/
/* $created: 11-Mar-2015 $ */

// #define fmin(a,b) ((a) < (b) ? (a) : (b))
// #define fmax(a,b) ((a) > (b) ? (a) : (b))
// #include "mex.h"
// #include "cuda.h"

__global__ void hank2patch_single(float* out,float* y,int sy,int sx,int sz,int firy, int firx)
{
    
    int idx = threadIdx.x+blockIdx.x*blockDim.x;
    int um=sy-firy+1,bm=sx-firx+1;
    int un=firy,bn=firx;
    int ii=0,jj=0,zz=0,jid=0,iid=0,si=0,sj=0,ci=0,cj=0,k=0,m=0;
    if ( idx < sy*sx*sz )
    {
        zz=(int)(idx/(sy*sx));
        k=idx%(sy*sx);
        ii=k%(sy);
        jj=(int)(k/sy);        
        out[ii + jj*sy + zz*sy*sx]=(float)0.0f;
        for (jid=0;jid<=jj;jid++)
        {
                if ((jid<bm) && ((jj-jid) <bn))
                {
                    si=jid*um;
                    sj=(jj-jid)*un;
                    for (iid=0;iid<=ii;iid++)
                    {
                        if ((iid<um) && ((ii-iid) <un))
                        {
                            ci=si+iid;
                            cj=sj+(ii-iid);
                            out[ii + jj*sy + zz*sy*sx]+=y[ci+cj*um*bm+zz*um*bm*un*bn];
                            m+=1;
                        }
                    }
                }
        }
        out[ii + jj*sy + zz*sy*sx]/=(float)m;
    }
}